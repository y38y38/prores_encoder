/**
 *
 * Copyright (c) 2020 Yuusuke Miyazaki
 *
 * This software is released under the MIT License.
 * http://opensource.org/licenses/mit-license.php
 *
 **/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
//#include <math.h>
#include <stdbool.h>
#include <pthread.h>

#include <fcntl.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#include <time.h>
#include <sys/time.h>

#include "config.h"
#include "dct.h"
#include "bitstream.h"
#include "vlc.h"
#include "slice.h"
#include "encoder.h"



struct thread_param {
	int thread_no;
	pthread_mutex_t  write_bitstream_my_mutex;
	pthread_mutex_t  *write_bitstream_next_mutex;
	int16_t y_slice[MAX_SLICE_DATA];
};



#define MACRO_BLOCK_Y_HORIZONTAL  (16)
#define MACRO_BLOCK_Y_VERTICAL    (16)

#define MACRO_BLOCK_422_C_HORIZONTAL  (8)
#define MACRO_BLOCK_422_C_VERTICAL    (16)

#define MAX_BITSTREAM_SIZE	(1073741824) //1M



static struct bitstream *write_bitstream;
//static uint8_t bitstream_buffer[MAX_BITSTREAM_SIZE];



//void start_write_bitstream(void);
//void frame_end_wait(void);

static uint16_t slice_size_table[MAX_SLICE_NUM];



uint32_t slice_num_max;



static uint32_t picture_size_offset_ = 0;


int32_t GetSliceNum(int32_t horizontal, int32_t vertical, int32_t sliceSize)
{
    int32_t mb_x_max = (horizontal + 15)  >> 4;
    int32_t mb_y_max = (vertical + 15) >> 4;


    int32_t slice_num_max_tmp;

    int32_t numMbsRemainingInRow = mb_x_max;
    int32_t number_of_slices_per_mb_row;
    int j = 0;

    do {
        while (numMbsRemainingInRow >= sliceSize) {
            j++;
            numMbsRemainingInRow  -=sliceSize;

        }
        sliceSize >>= 1;
    } while(numMbsRemainingInRow  > 0);

    number_of_slices_per_mb_row = j;

    slice_num_max_tmp = number_of_slices_per_mb_row * mb_y_max;
    return slice_num_max_tmp;

}
uint32_t GetEncodeHorizontal(int32_t horizontal)
{
    return ((horizontal + 15)  >> 4) << 4;

}
uint32_t GetEncodeVertical(int32_t vertical)
{
    return ((vertical + 15)  >> 4) << 4;
}



void set_picture_header(struct encoder_param* param)
{

    uint8_t picture_header_size = 0x8;
    setBit(write_bitstream, picture_header_size, 5);

    uint8_t reserved = 0x0;
    setBit(write_bitstream, reserved , 3);

    picture_size_offset_ = (getBitSize(write_bitstream)) >> 3 ;

    uint32_t picture_size = SET_DATA32(0);
    setByte(write_bitstream, (uint8_t*)&picture_size, 4);

    uint32_t slice_num = GetSliceNum(param->horizontal, param->vertical, param->slice_size_in_mb);
    uint16_t deprecated_number_of_slices =  SET_DATA16(slice_num);
    setByte(write_bitstream, (uint8_t*)&deprecated_number_of_slices , 0x2);


    uint8_t reserved2 = 0x0;
    setBit(write_bitstream, reserved2 , 2);

    uint8_t log2_desired_slice_size_in_mb;
    if (param->slice_size_in_mb == 1) {
        log2_desired_slice_size_in_mb = 0;
    } else if (param->slice_size_in_mb == 2) {
        log2_desired_slice_size_in_mb = 1;
    } else if (param->slice_size_in_mb == 4) {
        log2_desired_slice_size_in_mb = 2;
    } else {
        log2_desired_slice_size_in_mb = 3;
    }
    setBit(write_bitstream,log2_desired_slice_size_in_mb, 2);

    uint8_t reserved3 = 0x0;
    setBit(write_bitstream, reserved3 , 4);


}
void set_frame_header(struct encoder_param* param)
{
    uint16_t frame_header_size = SET_DATA16(0x94);
    setByte(write_bitstream, (uint8_t*)&frame_header_size, 0x2);

    uint8_t reserved = 0x0;
    setByte(write_bitstream, &reserved, 0x1);

    uint8_t bitstream_version = 0x0;
    setByte(write_bitstream, &bitstream_version, 0x1);


    uint32_t encoder_identifier = SET_DATA32(0x4c617663);
    setByte(write_bitstream, (uint8_t*)&encoder_identifier, 0x4);

    uint16_t horizontal_size = SET_DATA16(param->horizontal);
    setByte(write_bitstream, (uint8_t*)&horizontal_size , 0x2);

    uint16_t vertical_size = SET_DATA16(param->vertical);
    setByte(write_bitstream, (uint8_t*)&vertical_size, 0x2);


    uint8_t chroma_format;
    if (param->format_444 == true) {
        chroma_format = 0x3;
    } else {
        chroma_format = 0x2;
    }
    setBit(write_bitstream, chroma_format, 2);

    uint8_t reserved1 = 0x0;
    setBit(write_bitstream, reserved1, 2);

    uint8_t interlace_mode = 0;
    setBit(write_bitstream, interlace_mode, 2);

    uint8_t reserved2 = 0x0;
    setBit(write_bitstream, reserved2, 2);

    uint8_t aspect_ratio_information = 0;
    setBit(write_bitstream, aspect_ratio_information, 4);

    uint8_t frame_rate_code = 0;
    setBit(write_bitstream, frame_rate_code, 4);

    uint8_t color_primaries = 0x0;
    setByte(write_bitstream, &color_primaries, 1);

    uint8_t transfer_characteristic = 0x0;
    setByte(write_bitstream, &transfer_characteristic , 1);

    uint8_t matrix_coefficients = 0x2;
    setByte(write_bitstream, &matrix_coefficients, 1);


    uint8_t reserved3 = 0x4;
    setBit(write_bitstream, reserved3 , 4);

    //printf("1   %x %x\n", tmp_buf_byte_offset, tmp_buf[0x1b]);
    uint8_t alpha_channel_type = 0x0;
    setBit(write_bitstream, alpha_channel_type , 4);

    //printf("2   %x %x\n", tmp_buf_byte_offset, tmp_buf[0x1b]);
    uint8_t reserved4 = 0x0;
    setByte(write_bitstream, &reserved4 , 1);
    
    //printf("3   %x %x\n", tmp_buf_byte_offset, tmp_buf[0x1b]);
    uint8_t reserved5 = 0x0;
    setBit(write_bitstream, reserved5, 6);
    
    //printf("4   %x %x\n", tmp_buf_byte_offset, tmp_buf[0x1b]);
    uint8_t load_luma_quantization_matrix = 0x1;
    setBit(write_bitstream, load_luma_quantization_matrix, 1);

    //printf("5   %x %x\n", tmp_buf_byte_offset, tmp_buf[0x1b]);

    uint8_t load_chroma_quantization_matrix = 0x1;
    setBit(write_bitstream, load_chroma_quantization_matrix, 1);

    setByte(write_bitstream, param->luma_matrix, MATRIX_NUM );
    setByte(write_bitstream, param->chroma_matrix, MATRIX_NUM );


}

void setSliceTalbeFlush(uint16_t size, uint32_t offset) {
    uint16_t slice_size = SET_DATA16(size);
    setByteInOffset(write_bitstream, offset, (uint8_t*)&slice_size, 2);
    

}


int mbXFormSliceNo(struct Slice_cuda* slice_param, int slice_no)
{
	uint32_t mb_x_max = (slice_param->horizontal + 15) >>4;
	int horizontal_slice_num = mb_x_max /slice_param->slice_size_in_mb;

	int mb_x = (slice_no % horizontal_slice_num) * slice_param->slice_size_in_mb;
	return mb_x;
}
int mbYFormSliceNo(struct Slice_cuda* slice_param, int slice_no)
{
	uint32_t mb_x_max = (slice_param->horizontal + 15) >>4;
	int horizontal_slice_num = mb_x_max /slice_param->slice_size_in_mb;

	int mb_y = slice_no / horizontal_slice_num;
	return mb_y;
}


struct Slice_cuda h_slice_param_cuda;
//uint8_t h_qscale_table_cuda[MAX_SLICE_NUM];

void encode_slices(struct encoder_param * param)
{
    slice_num_max = GetSliceNum(param->horizontal, param->vertical, param->slice_size_in_mb);

    /* write dummy slice size table */
    int32_t i;
    uint32_t slice_size_table_offset = (getBitSize(write_bitstream)) >> 3 ;
    for (i = 0; i < slice_num_max ; i++) {
        uint16_t slice_size = 0x0;
        setByte(write_bitstream, (uint8_t*)&slice_size, 2);
    }

	memcpy(h_slice_param_cuda.luma_matrix, param->luma_matrix, BLOCK_IN_PIXEL);
    memcpy(h_slice_param_cuda.chroma_matrix, param->chroma_matrix, BLOCK_IN_PIXEL);
    h_slice_param_cuda.slice_size_in_mb= param->slice_size_in_mb;
    h_slice_param_cuda.horizontal= param->horizontal;
    h_slice_param_cuda.vertical= param->vertical;
    h_slice_param_cuda.format_444 = param->format_444;
	struct Slice_cuda * c_slice_param_cuda;

#ifndef HOST_ONLY
	hipError_t err;
	err = hipMalloc(&c_slice_param_cuda, sizeof(struct Slice_cuda));
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
	hipError_t err = hipMemcpy(c_slice_param_cuda, &h_slice_param_cuda, sizeof(struct Slice_cuda), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_slice_param_cuda = (struct Slice_cuda *)malloc(sizeof(struct Slice_cuda));
	if (c_slice_param_cuda == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memcpy(c_slice_param_cuda, &h_slice_param_cuda, sizeof(struct Slice_cuda));

#endif

	uint8_t *c_qscale_table;
#ifndef HOST_ONLY
	err = hipMalloc(&c_qscale_table, sizeof(uint8_t) * slice_num_max);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
	err = hipMemcpy(c_qscale_table, param->qscale_table, slice_num_max);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_qscale_table = (uint8_t*)malloc(sizeof(uint8_t) * slice_num_max);
	if (c_qscale_table == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memcpy(c_qscale_table, param->qscale_table, slice_num_max);
#endif

	uint16_t *c_y_data;
	int y_size = param->horizontal * param->vertical * sizeof(uint16_t);
#ifndef HOST_ONLY

	err = hipMalloc(&c_y_data, y_size);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
	err = hipMemcpy(c_y_data, param->y_data, y_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_y_data = (uint16_t*)malloc(y_size);
	if (c_y_data == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memcpy(c_y_data, param->y_data, y_size);
#endif

	uint16_t *c_cb_data;
	int cb_size;
	if (param->format_444 == true) {
		cb_size = y_size;
	} else {
		cb_size = y_size >> 1;
	}

#ifndef HOST_ONLY
	err = hipMalloc(&c_cb_data, cb_size);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
	err = hipMemcpy(c_cb_data, param->cb_data, cb_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_cb_data = (uint16_t*)malloc(cb_size);
	if (c_cb_data == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memcpy(c_cb_data, param->cb_data, cb_size);
#endif

	uint16_t *c_cr_data;
	int cr_size = cb_size;
#ifndef HOST_ONLY
	err = hipMalloc(&c_cr_data, cr_size);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
	err = hipMemcpy(c_cr_data, param->cb_data, cr_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_cr_data = (uint16_t*)malloc(cr_size);
	if (c_cr_data == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memcpy(c_cr_data, param->cr_data, cr_size);
#endif

	struct bitstream *c_bitstream;
	int bitstream_size = (sizeof(struct bitstream) + MAX_SLICE_BITSTREAM_SIZE) * slice_num_max;
#ifndef HOST_ONLY
	err = hipMalloc(&c_bitstream, bitstream_size);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	//printf("malloc size%d\n", bitstream_size);
	c_bitstream = (struct bitstream*)malloc(bitstream_size);
	if (c_bitstream == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memset(c_bitstream, 0x0, bitstream_size);
#endif


	uint16_t *c_slice_size_table;
	int slice_size_table_size = slice_num_max * sizeof(uint16_t);
#ifndef HOST_ONLY
	err = hipMalloc(&c_slice_size_table, slice_size_table_size);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_slice_size_table = (uint16_t*)malloc(slice_size_table_size);
	if (c_slice_size_table == NULL ) {
		printf("malloc error %d", __LINE__);
	}

#endif

	int16_t *c_working_buffer;//thread分のバッファを持つ必要あり。
	int working_buffer_size = (MAX_SLICE_DATA * 2) * slice_num_max;
#ifndef HOST_ONLY
	err = hipMalloc(&c_working_buffer, working_buffer_size);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_working_buffer = (int16_t*)malloc(working_buffer_size);
	if (c_working_buffer == NULL ) {
		printf("malloc error %d", __LINE__);
	}

#endif

	//int i;
	for(i = 0; i <slice_num_max;i++)  {
		encode_slice(i, c_slice_param_cuda, c_qscale_table, c_y_data, c_cb_data, c_cr_data, c_bitstream, c_slice_size_table, c_working_buffer);
	}




	memcpy(slice_size_table, c_slice_size_table, slice_size_table_size);
    for (i = 0; i < slice_num_max ; i++) {
		//printf("size=0x%x %x\n", slice_size_table[i], slice_size_table_size);
        setSliceTalbeFlush(slice_size_table[i], slice_size_table_offset + (i * 2));
		for(int j=0;j<128;j++) {
			//printf("%x ", c_bitstream[i].bitstream_buffer[j]);
		}
		//printf("\n%x\n", c_bitstream[i].bitstream_buffer);
		uint8_t *ptr = (uint8_t*)c_bitstream;
		struct bitstream * bptr = (struct bitstream*)(ptr + ((sizeof(struct bitstream) + MAX_SLICE_BITSTREAM_SIZE) * i));

		setByte(write_bitstream, bptr->bitstream_buffer, slice_size_table[i]);
    }

}

uint8_t *encode_frame(struct encoder_param* param, uint32_t *encode_frame_size)
{

	write_bitstream = (struct bitstream*)malloc(sizeof(struct bitstream) + MAX_BITSTREAM_SIZE);
	if (write_bitstream == NULL ) {
		printf("error malloc %d\n", __LINE__);
		return NULL;
	}
//	write_bitstream.bitstream_buffer = bitstream_buffer;
    initBitStream(write_bitstream);

    uint32_t frame_size_offset = getBitSize(write_bitstream) >> 3 ;
    uint32_t frame_size = SET_DATA32(0x0); 
    setByte(write_bitstream, (uint8_t*)&frame_size,4);

    uint32_t frame_identifier = SET_DATA32(0x69637066); //icpf


    setByte(write_bitstream, (uint8_t*)&frame_identifier,4);

    set_frame_header(param);
    uint32_t picture_size_offset = (getBitSize(write_bitstream)) >> 3 ;

    set_picture_header(param);

    encode_slices(param);
    uint32_t picture_end = (getBitSize(write_bitstream)) >>  3 ;

    uint32_t tmp  = picture_end - picture_size_offset;
    uint32_t picture_size = SET_DATA32(tmp);

    setByteInOffset(write_bitstream, picture_size_offset_, (uint8_t*)&picture_size, 4);


    uint8_t *ptr = getBitStream(write_bitstream, encode_frame_size);
    uint32_t frame_size_data = SET_DATA32(*encode_frame_size);
    setByteInOffset(write_bitstream, frame_size_offset, (uint8_t*)&frame_size_data , 4);
    return ptr;
}




void encoder_init(void)
{
	vlc_init();
	dct_init();
}




