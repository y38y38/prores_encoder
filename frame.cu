/**
 *
 * Copyright (c) 2020 Yuusuke Miyazaki
 *
 * This software is released under the MIT License.
 * http://opensource.org/licenses/mit-license.php
 *
 **/

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
//#include <math.h>
#include <stdbool.h>
#include <pthread.h>

#include <fcntl.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#include <time.h>
#include <sys/time.h>


#include "config.h"
#include "dct.h"
#include "bitstream.h"
#include "vlc.h"
#include "slice.h"
#include "encoder.h"



#define MACRO_BLOCK_Y_HORIZONTAL  (16)
#define MACRO_BLOCK_Y_VERTICAL    (16)

#define MACRO_BLOCK_422_C_HORIZONTAL  (8)
#define MACRO_BLOCK_422_C_VERTICAL    (16)

#define MAX_BITSTREAM_SIZE	(1073741824) //1M



static struct bitstream *write_bitstream;


static uint16_t slice_size_table[MAX_SLICE_NUM];

uint32_t slice_num_max;

static uint32_t picture_size_offset_ = 0;


int32_t GetSliceNum(int32_t horizontal, int32_t vertical, int32_t sliceSize)
{
    int32_t mb_x_max = (horizontal + 15)  >> 4;
    int32_t mb_y_max = (vertical + 15) >> 4;


    int32_t slice_num_max_tmp;

    int32_t numMbsRemainingInRow = mb_x_max;
    int32_t number_of_slices_per_mb_row;
    int j = 0;

    do {
        while (numMbsRemainingInRow >= sliceSize) {
            j++;
            numMbsRemainingInRow  -=sliceSize;

        }
        sliceSize >>= 1;
    } while(numMbsRemainingInRow  > 0);

    number_of_slices_per_mb_row = j;

    slice_num_max_tmp = number_of_slices_per_mb_row * mb_y_max;
    return slice_num_max_tmp;

}
uint32_t GetEncodeHorizontal(int32_t horizontal)
{
    return ((horizontal + 15)  >> 4) << 4;

}
uint32_t GetEncodeVertical(int32_t vertical)
{
    return ((vertical + 15)  >> 4) << 4;
}



void set_picture_header(struct encoder_param* param)
{

    uint8_t picture_header_size = 0x8;
    setBit(write_bitstream, picture_header_size, 5);

    uint8_t reserved = 0x0;
    setBit(write_bitstream, reserved , 3);

    picture_size_offset_ = (getBitSize(write_bitstream)) >> 3 ;

    uint32_t picture_size = SET_DATA32(0);
    setByte(write_bitstream, (uint8_t*)&picture_size, 4);

    uint32_t slice_num = GetSliceNum(param->horizontal, param->vertical, param->slice_size_in_mb);
    uint16_t deprecated_number_of_slices =  SET_DATA16(slice_num);
    setByte(write_bitstream, (uint8_t*)&deprecated_number_of_slices , 0x2);


    uint8_t reserved2 = 0x0;
    setBit(write_bitstream, reserved2 , 2);

    uint8_t log2_desired_slice_size_in_mb;
    if (param->slice_size_in_mb == 1) {
        log2_desired_slice_size_in_mb = 0;
    } else if (param->slice_size_in_mb == 2) {
        log2_desired_slice_size_in_mb = 1;
    } else if (param->slice_size_in_mb == 4) {
        log2_desired_slice_size_in_mb = 2;
    } else {
        log2_desired_slice_size_in_mb = 3;
    }
    setBit(write_bitstream,log2_desired_slice_size_in_mb, 2);

    uint8_t reserved3 = 0x0;
    setBit(write_bitstream, reserved3 , 4);


}
void set_frame_header(struct encoder_param* param)
{
    uint16_t frame_header_size = SET_DATA16(0x94);
    setByte(write_bitstream, (uint8_t*)&frame_header_size, 0x2);

    uint8_t reserved = 0x0;
    setByte(write_bitstream, &reserved, 0x1);

    uint8_t bitstream_version = 0x0;
    setByte(write_bitstream, &bitstream_version, 0x1);


    uint32_t encoder_identifier = SET_DATA32(0x4c617663);
    setByte(write_bitstream, (uint8_t*)&encoder_identifier, 0x4);

    uint16_t horizontal_size = SET_DATA16(param->horizontal);
    setByte(write_bitstream, (uint8_t*)&horizontal_size , 0x2);

    uint16_t vertical_size = SET_DATA16(param->vertical);
    setByte(write_bitstream, (uint8_t*)&vertical_size, 0x2);


    uint8_t chroma_format;
    if (param->format_444 == true) {
        chroma_format = 0x3;
    } else {
        chroma_format = 0x2;
    }
    setBit(write_bitstream, chroma_format, 2);

    uint8_t reserved1 = 0x0;
    setBit(write_bitstream, reserved1, 2);

    uint8_t interlace_mode = 0;
    setBit(write_bitstream, interlace_mode, 2);

    uint8_t reserved2 = 0x0;
    setBit(write_bitstream, reserved2, 2);

    uint8_t aspect_ratio_information = 0;
    setBit(write_bitstream, aspect_ratio_information, 4);

    uint8_t frame_rate_code = 0;
    setBit(write_bitstream, frame_rate_code, 4);

    uint8_t color_primaries = 0x0;
    setByte(write_bitstream, &color_primaries, 1);

    uint8_t transfer_characteristic = 0x0;
    setByte(write_bitstream, &transfer_characteristic , 1);

    uint8_t matrix_coefficients = 0x2;
    setByte(write_bitstream, &matrix_coefficients, 1);


    uint8_t reserved3 = 0x4;
    setBit(write_bitstream, reserved3 , 4);

    //printf("1   %x %x\n", tmp_buf_byte_offset, tmp_buf[0x1b]);
    uint8_t alpha_channel_type = 0x0;
    setBit(write_bitstream, alpha_channel_type , 4);

    //printf("2   %x %x\n", tmp_buf_byte_offset, tmp_buf[0x1b]);
    uint8_t reserved4 = 0x0;
    setByte(write_bitstream, &reserved4 , 1);
    
    //printf("3   %x %x\n", tmp_buf_byte_offset, tmp_buf[0x1b]);
    uint8_t reserved5 = 0x0;
    setBit(write_bitstream, reserved5, 6);
    
    //printf("4   %x %x\n", tmp_buf_byte_offset, tmp_buf[0x1b]);
    uint8_t load_luma_quantization_matrix = 0x1;
    setBit(write_bitstream, load_luma_quantization_matrix, 1);

    //printf("5   %x %x\n", tmp_buf_byte_offset, tmp_buf[0x1b]);

    uint8_t load_chroma_quantization_matrix = 0x1;
    setBit(write_bitstream, load_chroma_quantization_matrix, 1);

    setByte(write_bitstream, param->luma_matrix, MATRIX_NUM );
    setByte(write_bitstream, param->chroma_matrix, MATRIX_NUM );


}

void setSliceTalbeFlush(uint16_t size, uint32_t offset) {
    uint16_t slice_size = SET_DATA16(size);
    setByteInOffset(write_bitstream, offset, (uint8_t*)&slice_size, 2);
    

}



struct Slice_cuda h_slice_param_cuda;
//uint8_t h_qscale_table_cuda[MAX_SLICE_NUM];

double h_kc_value[KC_INDEX_MAX];

void encode_slices(struct encoder_param * param)
{
    slice_num_max = GetSliceNum(param->horizontal, param->vertical, param->slice_size_in_mb);

    /* write dummy slice size table */
    int32_t i;
    uint32_t slice_size_table_offset = (getBitSize(write_bitstream)) >> 3 ;
    for (i = 0; i < slice_num_max ; i++) {
        uint16_t slice_size = 0x0;
        setByte(write_bitstream, (uint8_t*)&slice_size, 2);
    }

	memcpy(h_slice_param_cuda.luma_matrix, param->luma_matrix, BLOCK_IN_PIXEL);
    memcpy(h_slice_param_cuda.chroma_matrix, param->chroma_matrix, BLOCK_IN_PIXEL);
    h_slice_param_cuda.slice_size_in_mb= param->slice_size_in_mb;
    h_slice_param_cuda.horizontal= param->horizontal;
    h_slice_param_cuda.vertical= param->vertical;
    h_slice_param_cuda.format_444 = param->format_444;
	h_slice_param_cuda.slice_num_max = slice_num_max;
	struct Slice_cuda * c_slice_param_cuda;

#ifdef CUDA_ENCODER
	hipError_t err;
	err = hipMalloc(&c_slice_param_cuda, sizeof(struct Slice_cuda));
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
	err = hipMemcpy(c_slice_param_cuda, &h_slice_param_cuda, sizeof(struct Slice_cuda), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_slice_param_cuda = (struct Slice_cuda *)malloc(sizeof(struct Slice_cuda));
	if (c_slice_param_cuda == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memcpy(c_slice_param_cuda, &h_slice_param_cuda, sizeof(struct Slice_cuda));

#endif

	uint8_t *c_qscale_table;
#ifdef CUDA_ENCODER
	err = hipMalloc(&c_qscale_table, sizeof(uint8_t) * slice_num_max);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
	err = hipMemcpy(c_qscale_table, param->qscale_table, slice_num_max, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_qscale_table = (uint8_t*)malloc(sizeof(uint8_t) * slice_num_max);
	if (c_qscale_table == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memcpy(c_qscale_table, param->qscale_table, slice_num_max);
#endif

	uint16_t *c_y_data;
	int y_size = param->horizontal * param->vertical * sizeof(uint16_t);
#ifdef CUDA_ENCODER

	err = hipMalloc(&c_y_data, y_size);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
	err = hipMemcpy(c_y_data, param->y_data, y_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_y_data = (uint16_t*)malloc(y_size);
	if (c_y_data == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memcpy(c_y_data, param->y_data, y_size);
#endif

	uint16_t *c_cb_data;
	int cb_size;
	if (param->format_444 == true) {
		cb_size = y_size;
	} else {
		cb_size = y_size >> 1;
	}

#ifdef CUDA_ENCODER
	err = hipMalloc(&c_cb_data, cb_size);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
	err = hipMemcpy(c_cb_data, param->cb_data, cb_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_cb_data = (uint16_t*)malloc(cb_size);
	if (c_cb_data == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memcpy(c_cb_data, param->cb_data, cb_size);
#endif

	uint16_t *c_cr_data;
	int cr_size = cb_size;
#ifdef CUDA_ENCODER
	err = hipMalloc(&c_cr_data, cr_size);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
	err = hipMemcpy(c_cr_data, param->cb_data, cr_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_cr_data = (uint16_t*)malloc(cr_size);
	if (c_cr_data == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memcpy(c_cr_data, param->cr_data, cr_size);
#endif

	struct bitstream *c_bitstream;
	int bitstream_size = (sizeof(struct bitstream) + MAX_SLICE_BITSTREAM_SIZE) * slice_num_max;
#ifdef CUDA_ENCODER
	err = hipMalloc(&c_bitstream, bitstream_size);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	//printf("malloc size%d\n", bitstream_size);
	c_bitstream = (struct bitstream*)malloc(bitstream_size);
	if (c_bitstream == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memset(c_bitstream, 0x0, bitstream_size);
#endif


	uint16_t *c_slice_size_table;
	int slice_size_table_size = slice_num_max * sizeof(uint16_t);
#ifdef CUDA_ENCODER
	err = hipMalloc(&c_slice_size_table, slice_size_table_size);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_slice_size_table = (uint16_t*)malloc(slice_size_table_size);
	if (c_slice_size_table == NULL ) {
		printf("malloc error %d", __LINE__);
	}

#endif

	int16_t *c_working_buffer;//thread分のバッファを持つ必要あり。
	int working_buffer_size = (MAX_SLICE_DATA * 2) * slice_num_max;
#ifdef CUDA_ENCODER
	err = hipMalloc(&c_working_buffer, working_buffer_size);
	if (err != hipSuccess) {
		printf("hipMemcpy error %d %d", __LINE__, err);
	}
#else
	c_working_buffer = (int16_t*)malloc(working_buffer_size);
	if (c_working_buffer == NULL ) {
		printf("malloc error %d", __LINE__);
	}

#endif


	double *c_kc_value;
	int kc_value_size = sizeof(double) * KC_INDEX_MAX;
#ifdef CUDA_ENCODER
	err = hipMalloc(&c_kc_value, kc_value_size);
	if (err != hipSuccess) {
		printf("hipMalloc error %d %d", __LINE__, err);
	}
	err = hipMemcpy(c_kc_value, h_kc_value, kc_value_size, hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("hipMalloc error %d %d", __LINE__, err);
	}
#else
	c_kc_value = (double*)malloc(kc_value_size);
	if (c_kc_value == NULL ) {
		printf("hipMalloc error %d", __LINE__);
	}
	memcpy(c_kc_value, h_kc_value, kc_value_size);
#endif	

#ifdef  CUDA_ENCODER
	i = 0;
#if 1
	int nElem = slice_num_max;
	dim3 block(1, 1);
	dim3 grid(slice_num_max);
	encode_slice<<<grid,block>>>(i, c_slice_param_cuda, c_qscale_table, c_y_data, c_cb_data, c_cr_data, c_bitstream, c_slice_size_table, c_working_buffer,c_kc_value);
#else
	encode_slice<<<1,1>>>(i, c_slice_param_cuda, c_qscale_table, c_y_data, c_cb_data, c_cr_data, c_bitstream, c_slice_size_table, c_working_buffer,c_kc_value);
#endif
#else
	//int i;
	for(i = 0; i <slice_num_max;i++)  {
		encode_slice(i, c_slice_param_cuda, c_qscale_table, c_y_data, c_cb_data, c_cr_data, c_bitstream, c_slice_size_table, c_working_buffer,c_kc_value);
	}
#endif



#ifdef CUDA_ENCODER
	err = hipMemcpy(slice_size_table, c_slice_size_table, slice_size_table_size,hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("hipMalloc error %d %d", __LINE__, err);
	}
	//printf("slice table %x\n", slice_size_table[0]);
#else
	memcpy(slice_size_table, c_slice_size_table, slice_size_table_size);
#endif

	struct bitstream* h_bitstream;
#ifdef CUDA_ENCODER
	h_bitstream = (struct bitstream*)malloc(bitstream_size);
	if (h_bitstream == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	err = hipMemcpy(h_bitstream, c_bitstream, bitstream_size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("hipMalloc error %d %d", __LINE__, err);
	}
#else
	h_bitstream = (struct bitstream*)malloc(bitstream_size);
	if (h_bitstream == NULL ) {
		printf("malloc error %d", __LINE__);
	}
	memcpy(h_bitstream, c_bitstream, bitstream_size);
#endif

    for (i = 0; i < slice_num_max ; i++) {
		//printf("size=0x%x %x\n", slice_size_table[i], slice_size_table_size);
        setSliceTalbeFlush(slice_size_table[i], slice_size_table_offset + (i * 2));
		for(int j=0;j<128;j++) {
			//printf("%x ", c_bitstream[i].bitstream_buffer[j]);
		}
		//printf("\n%x\n", c_bitstream[i].bitstream_buffer);
		uint8_t *ptr = (uint8_t*)h_bitstream;
		struct bitstream * bptr = (struct bitstream*)(ptr + ((sizeof(struct bitstream) + MAX_SLICE_BITSTREAM_SIZE) * i));

		setByte(write_bitstream, bptr->bitstream_buffer, slice_size_table[i]);
    }
#ifdef CUDA_ENCODER
	hipFree(c_slice_param_cuda);
	hipFree(c_qscale_table);
	hipFree(c_y_data);
	hipFree(c_cb_data);
	hipFree(c_cr_data);
	hipFree(c_bitstream);
	hipFree(c_slice_size_table);
	hipFree(c_working_buffer);
	hipFree(c_kc_value);
#else
	free(c_slice_param_cuda);
	free(c_qscale_table);
	free(c_y_data);
	free(c_cb_data);
	free(c_cr_data);
	free(c_bitstream);
	free(c_slice_size_table);
	free(c_working_buffer);
	free(c_kc_value);
#endif



}

uint8_t *encode_frame(struct encoder_param* param, uint32_t *encode_frame_size)
{

	write_bitstream = (struct bitstream*)malloc(sizeof(struct bitstream) + MAX_BITSTREAM_SIZE);
	if (write_bitstream == NULL ) {
		printf("error malloc %d\n", __LINE__);
		return NULL;
	}
//	write_bitstream.bitstream_buffer = bitstream_buffer;
    initBitStream(write_bitstream);

    uint32_t frame_size_offset = getBitSize(write_bitstream) >> 3 ;
    uint32_t frame_size = SET_DATA32(0x0); 
    setByte(write_bitstream, (uint8_t*)&frame_size,4);

    uint32_t frame_identifier = SET_DATA32(0x69637066); //icpf


    setByte(write_bitstream, (uint8_t*)&frame_identifier,4);

    set_frame_header(param);
    uint32_t picture_size_offset = (getBitSize(write_bitstream)) >> 3 ;

    set_picture_header(param);

    encode_slices(param);
    uint32_t picture_end = (getBitSize(write_bitstream)) >>  3 ;

    uint32_t tmp  = picture_end - picture_size_offset;
    uint32_t picture_size = SET_DATA32(tmp);

    setByteInOffset(write_bitstream, picture_size_offset_, (uint8_t*)&picture_size, 4);


    uint8_t *ptr = getBitStream(write_bitstream, encode_frame_size);
    uint32_t frame_size_data = SET_DATA32(*encode_frame_size);
    setByteInOffset(write_bitstream, frame_size_offset, (uint8_t*)&frame_size_data , 4);
    return ptr;
}




void encoder_init(void)
{
	//vlc_init();
	dct_init(h_kc_value);
}




