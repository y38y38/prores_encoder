#include "hip/hip_runtime.h"
/**
 *
 * Copyright (c) 2020 Yuusuke Miyazaki
 *
 * This software is released under the MIT License.
 * http://opensource.org/licenses/mit-license.php
 *
 **/
#include <stdio.h>
#include <stdint.h>
#include <math.h>

#include "config.h"
#include "prores.h"
#include "dct.h"
#include "dct_init.h"

void dct_init(double *kc_value)
{
    int h,v,x,y;
    for (v=0;v<MAX_Y;v++) {
        for (h=0;h<MAX_X;h++) {
		    for(y=0;y<MAX_Y;y++) {
        		for(x=0;x<MAX_X;x++) {
            		kc_value[GET_KC_INDEX(x,y,h,v)] = cos((M_PI * v * ((2.0 * y) + 1.0)) / 16.0) * cos((M_PI * h * ((2.0 * x) + 1.0)) / 16.0);
				}
			}
        }
    }
    return;
}

